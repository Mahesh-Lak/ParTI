
#include <hip/hip_runtime.h>
__global__ void spt_TTMKernel(
    double *Y_val, size_t Y_stride, size_t Y_nnz,
    const double *X_val, size_t X_nnz, const size_t *X_inds_m,
    const size_t *fiberidx_val, size_t fiberidx_len,
    const double *U_val, size_t U_nrows, size_t U_ncols, size_t U_stride,
    size_t block_offset
) {
    extern __shared__ double mem_pool[];

    const size_t tidx = threadIdx.x;
    const size_t tidy = threadIdx.y;
    const size_t i = (blockIdx.x + block_offset) * blockDim.x + tidx;
    //const size_t off = blockIdx.x * blockDim.x + tidx;
    size_t inz_begin, inz_end;
    if(i < Y_nnz) {
        inz_begin = fiberidx_val[i];
        inz_end = fiberidx_val[i+1];
    }
    __syncthreads();

    //double * const Y_shr = (double *) &mem_pool[tidx*Y_stride]; // size U_ncols
    double * const Y_shr = (double *) mem_pool; // size U_ncols
    if(i < Y_nnz && tidy < U_ncols) {
        Y_shr[tidx * Y_stride + tidy] = 0;
    }
    __syncthreads();

    if(i < Y_nnz && tidy < U_ncols) {
        for(size_t j = inz_begin; j < inz_end; ++j) {
            const size_t r = X_inds_m[j];
            Y_shr[tidx * Y_stride + tidy] += X_val[j] * U_val[r*U_stride + tidy];
        }
    }
    __syncthreads();

    if(i < Y_nnz && tidy < U_ncols) {
        Y_val[i*Y_stride + tidy] = Y_shr[tidx*Y_stride + tidy];
    }
    __syncthreads();
}
