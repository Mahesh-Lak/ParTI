
#include <hip/hip_runtime.h>
/*
    This file is part of SpTOL.

    SpTOL is free software: you can redistribute it and/or modify
    it under the terms of the GNU Lesser General Public License as
    published by the Free Software Foundation, either version 3 of
    the License, or (at your option) any later version.

    SpTOL is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU Lesser General Public
    License along with SpTOL.
    If not, see <http://www.gnu.org/licenses/>.
*/

__global__ void spt_TTMKernel(
    double *Y_val, size_t Y_stride, size_t Y_nnz,
    const double *X_val, size_t X_nnz, const size_t *X_inds_m,
    const size_t *fiberidx_val, size_t fiberidx_len,
    const double *U_val, size_t U_nrows, size_t U_ncols, size_t U_stride,
    size_t block_offset
) {
    extern __shared__ double mem_pool[];

    const size_t tidx = threadIdx.x;
    const size_t tidy = threadIdx.y;
    const size_t i = (blockIdx.x + block_offset) * blockDim.x + tidx;
    //const size_t off = blockIdx.x * blockDim.x + tidx;
    size_t inz_begin, inz_end;
    if(i < Y_nnz) {
        inz_begin = fiberidx_val[i];
        inz_end = fiberidx_val[i+1];
    }
    __syncthreads();

    //double * const Y_shr = (double *) &mem_pool[tidx*Y_stride]; // size U_ncols
    double * const Y_shr = (double *) mem_pool; // size U_ncols
    if(i < Y_nnz && tidy < U_ncols) {
        Y_shr[tidx * Y_stride + tidy] = 0;
    }
    __syncthreads();

    if(i < Y_nnz && tidy < U_ncols) {
        for(size_t j = inz_begin; j < inz_end; ++j) {
            const size_t r = X_inds_m[j];
            Y_shr[tidx * Y_stride + tidy] += X_val[j] * U_val[r*U_stride + tidy];
        }
    }
    __syncthreads();

    if(i < Y_nnz && tidy < U_ncols) {
        Y_val[i*Y_stride + tidy] = Y_shr[tidx*Y_stride + tidy];
    }
    __syncthreads();
}
