#include "hip/hip_runtime.h"
/*
    This file is part of ParTI!.

    ParTI! is free software: you can redistribute it and/or modify
    it under the terms of the GNU Lesser General Public License as
    published by the Free Software Foundation, either version 3 of
    the License, or (at your option) any later version.

    ParTI! is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU Lesser General Public
    License along with ParTI!.
    If not, see <http://www.gnu.org/licenses/>.
*/

#include <ParTI.h>
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include "sptensor.h"
#include "mmul_cuda_kernels.h"


/**
 * CUDA parallelized Sparse tensor times a dense matrix (SpTTM)
 * @param[out] Y    the result of X*U, should be uninitialized
 * @param[in]  X    the sparse tensor input X
 * @param[in]  U    the dense matrix input U
 * @param      mode the mode on which the multiplication is done on
 *
 * This function will sort Y with `sptSparseTensorSortIndexAtMode`
 * automatically, this operation can be undone with `sptSparseTensorSortIndex`
 * if you need to access raw data.
 * Anyway, you do not have to take this side-effect into consideration if you
 * do not need to access raw data.
 */
int sptCudaSparseTensorMulMatrix(
    sptSemiSparseTensor *Y,
    sptSparseTensor *X,
    const sptMatrix *U,
    size_t mode
) {
    int result;
    size_t *ind_buf;
    size_t m;
    sptSizeVector fiberidx;
    if(mode >= X->nmodes) {
        spt_CheckError(SPTERR_SHAPE_MISMATCH, "CUDA SpTns * Mtx", "shape mismatch");
    }
    if(X->ndims[mode] != U->nrows) {
        spt_CheckError(SPTERR_SHAPE_MISMATCH, "CUDA SpTns * Mtx", "shape mismatch");
    }
    sptSparseTensorSortIndexAtMode(X, mode, 0);
    ind_buf = new size_t[X->nmodes * sizeof *ind_buf];
    for(m = 0; m < X->nmodes; ++m) {
        ind_buf[m] = X->ndims[m];
    }
    ind_buf[mode] = U->ncols;
    result = sptNewSemiSparseTensor(Y, X->nmodes, mode, ind_buf);
    delete[] ind_buf;
    spt_CheckError(result, "CUDA SpTns * Mtx", NULL);
    sptSemiSparseTensorSetIndices(Y, &fiberidx, X);

    sptScalar *Y_val = NULL;
    result = hipMalloc((void **) &Y_val, Y->nnz * Y->stride * sizeof (sptScalar));
    spt_CheckCudaError(result != 0, "CUDA SpTns * Mtx");
    // jli: Add memset to Y.
    hipMemset(Y_val, 0, Y->nnz * Y->stride * sizeof (sptScalar));
    sptScalar *X_val = NULL;
    result = hipMalloc((void **) &X_val, X->nnz * sizeof (sptScalar));
    spt_CheckCudaError(result != 0, "CUDA SpTns * Mtx");
    hipMemcpy(X_val, X->values.data, X->nnz * sizeof (sptScalar), hipMemcpyHostToDevice);
    size_t *X_inds_m = NULL;
    result = hipMalloc((void **) &X_inds_m, X->nnz * sizeof (size_t));
    spt_CheckCudaError(result != 0, "CUDA SpTns * Mtx");
    hipMemcpy(X_inds_m, X->inds[mode].data, X->nnz * sizeof (size_t), hipMemcpyHostToDevice);
    sptScalar *U_val = NULL;
    result = hipMalloc((void **) &U_val, U->nrows * U->stride * sizeof (sptScalar));
    spt_CheckCudaError(result != 0, "CUDA SpTns * Mtx");
    hipMemcpy(U_val, U->values, U->nrows * U->stride * sizeof (sptScalar), hipMemcpyHostToDevice);
    size_t *fiberidx_val = NULL;
    result = hipMalloc((void **) &fiberidx_val, fiberidx.len * sizeof (size_t));
    spt_CheckCudaError(result != 0, "CUDA SpTns * Mtx");
    hipMemcpy(fiberidx_val, fiberidx.data, fiberidx.len * sizeof (size_t), hipMemcpyHostToDevice);

    const char *env_PARTI_TTM_KERNEL = getenv("PARTI_TTM_KERNEL");
    const bool use_naive_kernel = env_PARTI_TTM_KERNEL && !strcmp(env_PARTI_TTM_KERNEL, "naive");

    const size_t max_nblocks = 32768;
    const size_t max_nthreads = 1024;
    // size_t sharedMem = (Y->ndims[mode] + X->ndims[mode])*sizeof (sptScalar) + X->ndims[mode]*sizeof (size_t);
    const char *env_PARTI_TTM_NTHREADS = getenv("PARTI_TTM_NTHREADS");
    size_t nthreadsX = 32;
    if(env_PARTI_TTM_NTHREADS) {
        sscanf(env_PARTI_TTM_NTHREADS, "%zu", &nthreadsX);
    }
    size_t sharedMem = nthreadsX * Y->stride * sizeof (sptScalar);

    size_t all_nblocks = Y->nnz % nthreadsX == 0 ? Y->nnz / nthreadsX : Y->nnz / nthreadsX + 1;
    assert(U->ncols < max_nthreads);
    dim3 dimBlock(nthreadsX, U->ncols);
    // size_t nblocks = Y->nnz < max_nblocks ? Y->nnz : max_nblocks;

    if(!use_naive_kernel) {
        fprintf(stderr, "[CUDA SpTns * Mtx] spt_TTMKernel<<<%zu, (%u, %u), %zu>>>\n", all_nblocks, dimBlock.x, dimBlock.y, sharedMem);
    } else {
        fprintf(stderr, "[CUDA SpTns * Mtx] spt_TTMNaiveKernel<<<%zu, (%u, %u), 0>>>\n", all_nblocks, dimBlock.x, dimBlock.y);
    }

    sptTimer timer;
    sptNewTimer(&timer, 0);
    sptStartTimer(timer);

    for(size_t block_offset = 0; block_offset < all_nblocks; block_offset += max_nblocks) {
        size_t nblocks = all_nblocks - block_offset;
        if(nblocks > max_nblocks) {
            nblocks = max_nblocks;
        }
        if(!use_naive_kernel) {
            spt_TTMKernel<<<nblocks, dimBlock, sharedMem>>>(
                Y_val, Y->stride, Y->nnz,
                X_val, X->nnz, X_inds_m,
                fiberidx_val, fiberidx.len,
                U_val, U->nrows, U->ncols, U->stride,
                block_offset
            );
        } else {
            spt_TTMNaiveKernel<<<nblocks, dimBlock>>>(
                Y_val, Y->stride, Y->nnz,
                X_val, X->nnz, X_inds_m,
                fiberidx_val, fiberidx.len,
                U_val, U->nrows, U->ncols, U->stride,
                block_offset
            );
        }
        result = hipDeviceSynchronize();
        spt_CheckCudaError(result != 0, "CUDA SpTns * Mtx kernel");
    }

    sptStopTimer(timer);
    sptPrintElapsedTime(timer, "CUDA SpTns * Mtx");
    sptFreeTimer(timer);

    hipMemcpy(Y->values.values, Y_val, Y->nnz * Y->stride * sizeof (sptScalar), hipMemcpyDeviceToHost);
    result = hipFree(fiberidx_val);
    spt_CheckCudaError(result != 0, "CUDA SpTns * Mtx");
    result = hipFree(U_val);
    spt_CheckCudaError(result != 0, "CUDA SpTns * Mtx");
    result = hipFree(X_inds_m);
    spt_CheckCudaError(result != 0, "CUDA SpTns * Mtx");
    result = hipFree(X_val);
    spt_CheckCudaError(result != 0, "CUDA SpTns * Mtx");
    result = hipFree(Y_val);
    spt_CheckCudaError(result != 0, "CUDA SpTns * Mtx");
    sptFreeSizeVector(&fiberidx);

    return 0;
}



/**
 * CUDA parallelized Sparse tensor times a dense matrix (SpTTM)
 * @param[out] Y    the result of X*U, should be uninitialized
 * @param[in]  X    the sparse tensor input X
 * @param[in]  U    the dense matrix input U
 * @param      mode the mode on which the multiplication is done on
 *
 * This function will sort Y with `sptSparseTensorSortIndexAtMode`
 * automatically, this operation can be undone with `sptSparseTensorSortIndex`
 * if you need to access raw data.
 * Anyway, you do not have to take this side-effect into consideration if you
 * do not need to access raw data.
 */
int sptCudaSparseTensorMulMatrixOneKernel(
    sptSemiSparseTensor *Y,
    sptSparseTensor *X,
    const sptMatrix *U,
    size_t mode,
    size_t const impl_num,
    size_t const smen_size) 
{
    int result;
    size_t *ind_buf;
    size_t m;
    sptSizeVector fiberidx;
    if(mode >= X->nmodes) {
        spt_CheckError(SPTERR_SHAPE_MISMATCH, "CUDA SpTns * Mtx", "shape mismatch");
    }
    if(X->ndims[mode] != U->nrows) {
        spt_CheckError(SPTERR_SHAPE_MISMATCH, "CUDA SpTns * Mtx", "shape mismatch");
    }
    sptSparseTensorSortIndexAtMode(X, mode, 0);
    ind_buf = new size_t[X->nmodes * sizeof *ind_buf];
    for(m = 0; m < X->nmodes; ++m) {
        ind_buf[m] = X->ndims[m];
    }
    ind_buf[mode] = U->ncols;
    result = sptNewSemiSparseTensor(Y, X->nmodes, mode, ind_buf);
    delete[] ind_buf;
    spt_CheckError(result, "CUDA SpTns * Mtx", NULL);
    sptSemiSparseTensorSetIndices(Y, &fiberidx, X);

    double flen = (double)X->nnz / fiberidx.len;
    size_t tmp_flen = (fiberidx.data[1] - fiberidx.data[0]) - flen;
    double fvar = tmp_flen * tmp_flen;
    for(size_t i=1; i<fiberidx.len - 1; ++i) {
        tmp_flen = (fiberidx.data[i+1] - fiberidx.data[i]) - flen;
        fvar += tmp_flen * tmp_flen;
    }
    tmp_flen = (X->nnz - fiberidx.data[fiberidx.len - 1]) - flen;
    fvar += tmp_flen * tmp_flen;
    fvar = sqrt(fvar);
    printf("nfibs: %zu, flen: %.2f, fvar: %.2f\n", fiberidx.len, flen, fvar);

    sptScalar *Y_val = NULL;
    result = hipMalloc((void **) &Y_val, Y->nnz * Y->stride * sizeof (sptScalar));
    spt_CheckCudaError(result != 0, "CUDA SpTns * Mtx");
    // jli: Add memset to Y.
    hipMemset(Y_val, 0, Y->nnz * Y->stride * sizeof (sptScalar));
    sptScalar *X_val = NULL;
    result = hipMalloc((void **) &X_val, X->nnz * sizeof (sptScalar));
    spt_CheckCudaError(result != 0, "CUDA SpTns * Mtx");
    hipMemcpy(X_val, X->values.data, X->nnz * sizeof (sptScalar), hipMemcpyHostToDevice);
    size_t *X_inds_m = NULL;
    result = hipMalloc((void **) &X_inds_m, X->nnz * sizeof (size_t));
    spt_CheckCudaError(result != 0, "CUDA SpTns * Mtx");
    hipMemcpy(X_inds_m, X->inds[mode].data, X->nnz * sizeof (size_t), hipMemcpyHostToDevice);
    sptScalar *U_val = NULL;
    result = hipMalloc((void **) &U_val, U->nrows * U->stride * sizeof (sptScalar));
    spt_CheckCudaError(result != 0, "CUDA SpTns * Mtx");
    hipMemcpy(U_val, U->values, U->nrows * U->stride * sizeof (sptScalar), hipMemcpyHostToDevice);
    size_t *fiberidx_val = NULL;
    result = hipMalloc((void **) &fiberidx_val, fiberidx.len * sizeof (size_t));
    spt_CheckCudaError(result != 0, "CUDA SpTns * Mtx");
    hipMemcpy(fiberidx_val, fiberidx.data, fiberidx.len * sizeof (size_t), hipMemcpyHostToDevice);

    const size_t max_nblocks = 32768;
    const size_t max_nthreads_per_block = 256;
    size_t max_nthreadsy = 16;

    size_t nthreadsx = 1;
    size_t nthreadsy = 1;
    size_t all_nblocks = 0;
    size_t nblocks = 0;

    const char *env_PARTI_TTM_NTHREADS = getenv("PARTI_TTM_NTHREADS");

    switch(impl_num) {
    // case 1:
    case 11: // Naive, 1D
        if(Y->nnz < max_nthreads_per_block) {
            nthreadsx = Y->nnz;
            nblocks = 1;
        } else {
            nthreadsx = max_nthreads_per_block;
            all_nblocks = (Y->nnz + nthreadsx -1) / nthreadsx;
            if(all_nblocks < max_nblocks) {
                nblocks = all_nblocks;
            } else {
                nblocks = max_nblocks;
            }
        }
        break;
    case 12:
        if(U->ncols <= max_nthreadsy)
            nthreadsy = U->ncols;
        else
            nthreadsy = max_nthreadsy;
        nthreadsx = max_nthreads_per_block / nthreadsy;

        if(Y->nnz < nthreadsx) {
            nthreadsx = Y->nnz;
            nblocks = 1;
        } else {
            all_nblocks = (Y->nnz + nthreadsx -1) / nthreadsx;
            if(all_nblocks < max_nblocks) {
                nblocks = all_nblocks;
            } else {
                nblocks = max_nblocks;
            }
        }
        break;
    case 13:
    case 14:
        if(U->ncols <= max_nthreadsy)
            nthreadsx = U->ncols;
        else
            nthreadsx = max_nthreadsy;
        nthreadsy = max_nthreads_per_block / nthreadsx;

        if(Y->nnz < nthreadsy) {
            nthreadsy = Y->nnz;
            nblocks = 1;
        } else {
            all_nblocks = (Y->nnz + nthreadsy -1) / nthreadsy;
            if(all_nblocks < max_nblocks) {
                nblocks = all_nblocks;
            } else {
                nblocks = max_nblocks;
            }
        }
        break;
    case 15:
        if(U->ncols <= max_nthreadsy)
            nthreadsx = U->ncols;
        else
            nthreadsx = max_nthreadsy;
        nthreadsy = max_nthreads_per_block / nthreadsx;

        if(Y->nnz < nthreadsy) {
            nthreadsy = Y->nnz;
            nblocks = 1;
        } else {
            all_nblocks = (Y->nnz + nthreadsy -1) / nthreadsy;
            if(all_nblocks < max_nblocks) {
                nblocks = all_nblocks;
            } else {
                nblocks = max_nblocks;
            }
        }
        assert(smen_size >= nthreadsx * nthreadsy * sizeof (sptScalar));
        break;
    }
    dim3 dimBlock(nthreadsx, nthreadsy);
    printf("all_nblocks: %zu, nthreadsx: %zu, nthreadsy: %zu\n", all_nblocks, nthreadsx, nthreadsy);

    sptTimer timer;
    sptNewTimer(&timer, 0);
    sptStartTimer(timer);


    switch(impl_num) {
    // case 1:
    case 11: // Naive
        printf("[CUDA SpTns * Mtx] spt_TTMNnzKernel<<<%zu, (%zu, %zu)>>>\n", nblocks, nthreadsx, nthreadsy);
        spt_TTMNnzKernel<<<nblocks, dimBlock>>>(
            Y_val, Y->stride, Y->nnz,
            X_val, X->nnz, X_inds_m,
            fiberidx_val, fiberidx.len,
            U_val, U->nrows, U->ncols, U->stride);
        break;
    case 12:  
        printf("[CUDA SpTns * Mtx] spt_TTMNnzRankKernel<<<%zu, (%zu, %zu)>>>\n", nblocks, nthreadsx, nthreadsy);
        spt_TTMNnzRankKernel<<<nblocks, dimBlock>>>(
            Y_val, Y->stride, Y->nnz,
            X_val, X->nnz, X_inds_m,
            fiberidx_val, fiberidx.len,
            U_val, U->nrows, U->ncols, U->stride);
        break; 
    case 13:  
        printf("[CUDA SpTns * Mtx] spt_TTMRankNnzKernel<<<%zu, (%zu, %zu)>>>\n", nblocks, nthreadsx, nthreadsy);
        spt_TTMRankNnzKernel<<<nblocks, dimBlock>>>(
            Y_val, Y->stride, Y->nnz,
            X_val, X->nnz, X_inds_m,
            fiberidx_val, fiberidx.len,
            U_val, U->nrows, U->ncols, U->stride);
        break; 
    case 14:  
        printf("[CUDA SpTns * Mtx] spt_TTMRankRBNnzKernel<<<%zu, (%zu, %zu)>>>\n", nblocks, nthreadsx, nthreadsy);
        spt_TTMRankRBNnzKernel<<<nblocks, dimBlock>>>(
            Y_val, Y->stride, Y->nnz,
            X_val, X->nnz, X_inds_m,
            fiberidx_val, fiberidx.len,
            U_val, U->nrows, U->ncols, U->stride);
        break; 
    case 15:  
        printf("[CUDA SpTns * Mtx] spt_TTMRankRBNnzKernelSM<<<%zu, (%zu, %zu), %zu>>>\n", nblocks, nthreadsx, nthreadsy, smen_size);
        spt_TTMRankRBNnzKernelSM<<<nblocks, dimBlock, smen_size>>>(
            Y_val, Y->stride, Y->nnz,
            X_val, X->nnz, X_inds_m,
            fiberidx_val, fiberidx.len,
            U_val, U->nrows, U->ncols, U->stride);
        break; 
    }
    result = hipDeviceSynchronize();
    spt_CheckCudaError(result != 0, "CUDA SpTns * Mtx kernel");

    sptStopTimer(timer);
    sptPrintElapsedTime(timer, "CUDA SpTns * Mtx");
    sptFreeTimer(timer);

    hipMemcpy(Y->values.values, Y_val, Y->nnz * Y->stride * sizeof (sptScalar), hipMemcpyDeviceToHost);
    result = hipFree(fiberidx_val);
    spt_CheckCudaError(result != 0, "CUDA SpTns * Mtx");
    result = hipFree(U_val);
    spt_CheckCudaError(result != 0, "CUDA SpTns * Mtx");
    result = hipFree(X_inds_m);
    spt_CheckCudaError(result != 0, "CUDA SpTns * Mtx");
    result = hipFree(X_val);
    spt_CheckCudaError(result != 0, "CUDA SpTns * Mtx");
    result = hipFree(Y_val);
    spt_CheckCudaError(result != 0, "CUDA SpTns * Mtx");
    sptFreeSizeVector(&fiberidx);

    return 0;
}
