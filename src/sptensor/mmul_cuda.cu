#include "hip/hip_runtime.h"
#include <SpTOL.h>

__global__ static void spt_TTMKernel(
    sptScalar *Y_val, size_t Y_stride, size_t Y_nnz,
    const sptScalar *X_val, size_t X_nnz, size_t *X_inds_m,
    size_t *fiberidx_val, size_t fiberidx_len,
    const sptScalar *U_val, size_t U_nrows, size_t U_ncols, size_t U_stride
) {
    __shared__ char *mem_pool;
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    sptScalar *const Y_shr = (sptScalar *) &mem_pool[0]; // size U_ncols
    sptScalar *const X_shr = (sptScalar *) &mem_pool[U_ncols * sizeof (sptScalar)]; // size U_nrows
    size_t *const r_shr = (size_t *) &mem_pool[(U_ncols+U_nrows) * sizeof (sptScalar)]; // size U_nrows
    if(tid < Y_nnz) {
        size_t inz_begin = fiberidx_val[tid];
        size_t inz_end = fiberidx_val[tid+1];
        size_t j, k;
        for(k = 0; k < U_ncols; ++k) {
            Y_val[j] = 0;
        }
        for(j = 0; j < inz_end-inz_begin; ++j) {
            X_shr[j] = X_val[j+inz_begin];
        }
        for(j = 0; j < inz_end-inz_begin; ++j) {
            r_shr[j] = X_inds_m[j+inz_begin];
        }
        for(k = 0; k < U_ncols; ++k) {
            for(j = 0; j < inz_end-inz_begin; ++j) {
                Y_shr[k] += X_shr[j] * U_val[r_shr[j]*U_stride + k];
            }
        }
        for(k = 0; k < U_ncols; ++k) {
            Y_val[tid*Y_stride + k] = Y_shr[k];
        }
    }
}

static size_t spt_GetBlockCount(size_t threads) {
    return (threads / 256) + ((threads & 255) != 0);
}

int sptCudaSparseTensorMulMatrix(
    sptSemiSparseTensor *Y,
    sptSparseTensor *X,
    const sptMatrix *U,
    size_t mode
) {
    int result;
    size_t *ind_buf;
    size_t m;
    sptSizeVector fiberidx;
    if(mode >= X->nmodes) {
        return -1;
    }
    if(X->ndims[mode] != U->nrows) {
        return -1;
    }
    if(X->sortkey != mode) {
        sptSparseTensorSortIndexAtMode(X, mode);
    }
    ind_buf = new size_t[X->nmodes * sizeof *ind_buf];
    if(!ind_buf) {
        return -1;
    }
    for(m = 0; m < X->nmodes; ++m) {
        ind_buf[m] = X->ndims[m];
    }
    ind_buf[mode] = U->ncols;
    result = sptNewSemiSparseTensor(Y, X->nmodes, mode, ind_buf);
    delete[] ind_buf;
    if(result) {
        return result;
    }
    sptSemiSparseTensorSetIndices(Y, &fiberidx, X);

    size_t blocks_count = spt_GetBlockCount(Y->nnz);
    sptScalar *Y_val = NULL;
    result = hipMalloc((void **) &Y_val, Y->nnz * Y->stride * sizeof (sptScalar));
    if(result != 0) {
        return result; // TODO: map error code?
    }
    sptScalar *X_val = NULL;
    result = hipMalloc((void **) &X_val, X->nnz * sizeof (sptScalar));
    if(result != 0) {
        return result; // TODO: map error code?
    }
    hipMemcpy(X_val, X->values.data, X->nnz * sizeof (sptScalar), hipMemcpyHostToDevice);
    size_t *X_inds_m = NULL;
    result = hipMalloc((void **) &X_inds_m, X->nnz * sizeof (size_t));
    if(result != 0) {
        return result; // TODO: map error code?
    }
    hipMemcpy(X_inds_m, X->inds[mode].data, X->nnz * sizeof (size_t), hipMemcpyHostToDevice);
    sptScalar *U_val = NULL;
    result = hipMalloc((void **) &U_val, U->nrows * U->stride * sizeof (sptScalar));
    if(result != 0) {
        return result;
    }
    hipMemcpy(U_val, U->values, U->nrows * U->stride * sizeof (sptScalar), hipMemcpyHostToDevice);
    size_t *fiberidx_val = NULL;
    result = hipMalloc((void **) &fiberidx_val, fiberidx.len * sizeof (size_t));
    if(result != 0) {
        return result;
    }
    hipMemcpy(fiberidx_val, fiberidx.data, fiberidx.len * sizeof (size_t), hipMemcpyHostToDevice);

    size_t sharedMem = (Y->ndims[mode] + X->ndims[mode])*sizeof (sptScalar) + X->ndims[mode]*sizeof (size_t);

    spt_TTMKernel<<<blocks_count, 256, sharedMem>>>(
        Y_val, Y->stride, Y->nnz,
        X_val, X->nnz, X_inds_m,
        fiberidx_val, fiberidx.len,
        U_val, U->nrows, U->ncols, U->stride
    );

    hipMemcpy(Y->values.values, Y_val, Y->nnz * Y->stride * sizeof (sptScalar), hipMemcpyDeviceToHost);
    hipFree(fiberidx_val); hipFree(U_val); hipFree(X_inds_m); hipFree(X_val); hipFree(Y_val);
    sptFreeSizeVector(&fiberidx);

    return 0;
}
