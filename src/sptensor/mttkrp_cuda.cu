/*
    This file is part of ParTI!.

    ParTI! is free software: you can redistribute it and/or modify
    it under the terms of the GNU Lesser General Public License as
    published by the Free Software Foundation, either version 3 of
    the License, or (at your option) any later version.

    ParTI! is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU Lesser General Public
    License along with ParTI!.
    If not, see <http://www.gnu.org/licenses/>.
*/

#include <ParTI.h>
#include "sptensor.h"
#include "../cudawrap.h"
#include "mttkrp_cuda_kernels.h"



/**
 * CUDA parallelized Matriced sparse tensor times a sequence of dense matrix Khatri-Rao products (MTTKRP) on a specified mode
 * @param[out] mats[nmodes]    the result of MTTKRP, a dense matrix, with size
 * ndims[mode] * R
 * @param[in]  X    the sparse tensor input X
 * @param[in]  mats    (N+1) dense matrices, with mats[nmodes] as temporary
 * @param[in]  mats_order    the order of the Khatri-Rao products
 * @param[in]  mode   the mode on which the MTTKRP is performed
 * @param[in]  scratch an temporary array to store intermediate results, space assigned before this function
 *
 * This function uses support arbitrary-order sparse tensors with Khatri-Rao
 * products of dense factor matrices, the output is the updated dense matrix for the "mode".
 * In this version, atomic function to lock the global reduction and a large
 * scratch is used to maximize parallelism. (To be optimized)
 */
int sptCudaMTTKRP(
    sptSparseTensor const * const X,
    sptMatrix ** const mats,     // mats[nmodes] as temporary space.
    size_t * const mats_order,    // Correspond to the mode order of X.
    size_t const mode,
    size_t const impl_num) 
{
    size_t const nmodes = X->nmodes;
    size_t const nnz = X->nnz;
    size_t const * const ndims = X->ndims;
    size_t const R = mats[mode]->ncols;
    size_t const stride = mats[mode]->stride;
    int result;

    double time_h2d, time_exe, time_d2h;
    double gbw_h2d, gflops_exe, gbw_d2h;
    sptTimer timer;
    sptNewTimer(&timer, 0);

    /* Check the mats. */
    for(size_t i=0; i<nmodes; ++i) {
        if(mats[i]->ncols != mats[nmodes]->ncols) {
            spt_CheckError(SPTERR_SHAPE_MISMATCH, "CUDA SpTns MTTKRP", "mats[i]->cols != mats[nmodes]->ncols");
        }
        if(mats[i]->nrows != ndims[i]) {
            spt_CheckError(SPTERR_SHAPE_MISMATCH, "CUDA SpTns MTTKRP", "mats[i]->nrows != ndims[i]");
        }
    }


    /* Transfer tensor and matrices */
    /* dev_mats_order: 1st gpu. */
    size_t * dev_mats_order;
    /* dev_Xndims: 1st gpu. */
    size_t * dev_Xndims;
    /* dev_Xvals: 1st gpu. */
    sptScalar * dev_Xvals;
    /* Xinds_header: 1st cpu, 2nd cpu (ghost pointers) */
    size_t ** Xinds_header = new size_t *[nmodes];
    /* dev_Xinds: 1st gpu, 2nd gpu. */
    size_t ** dev_Xinds;
    /* mats_header: 1st cpu, 2nd cpu (ghost pointers) */
    sptScalar ** mats_header = new sptScalar *[nmodes+1];
    /* lengths: 1st cpu, store the lengths of mats */
    size_t * const lengths = new size_t[nmodes+1];
    /* dev_mats: 1st gpu, 2nd gpu. */
    sptScalar ** dev_mats;
    /* dev_scratch: 1st gpu. */
    sptScalar * dev_scratch;
    /* the pointer to dev_mats[nmodes] */
    sptScalar *dev_part_prod;  
    size_t dev_mem_size = 0;
    size_t dev_flops = 2 * nnz * R + (nmodes - 1) * R;


    sptStartTimer(timer);

    /* dev_mats_order */
    result = sptCudaDuplicateMemory(&dev_mats_order, mats_order, nmodes * sizeof (size_t), hipMemcpyHostToDevice);
    spt_CheckCudaError(result != 0, "CUDA SpTns SpltMTTKRP");
    dev_mem_size += nmodes * sizeof (size_t);

    /* dev_Xndims */
    result = sptCudaDuplicateMemory(&dev_Xndims, ndims, nmodes * sizeof (size_t), hipMemcpyHostToDevice);
    spt_CheckCudaError(result != 0, "CUDA SpTns SpltMTTKRP");
    dev_mem_size += nmodes * sizeof (size_t);

    /* dev_Xvals */
    result = sptCudaDuplicateMemory(&dev_Xvals, X->values.data, nnz * sizeof (sptScalar), hipMemcpyHostToDevice);
    spt_CheckCudaError(result != 0, "CUDA SpTns SpltMTTKRP");
    dev_mem_size += nnz * sizeof (sptScalar);

    /* Xinds_header */
    for(size_t m = 0; m < nmodes; ++m) {
        Xinds_header[m] = X->inds[m].data;
    }
    /* dev_Xinds */
    result = sptCudaDuplicateMemoryIndirect(&dev_Xinds, Xinds_header, nmodes, nnz, hipMemcpyHostToDevice);
    spt_CheckCudaError(result != 0, "CUDA SpTns SpltMTTKRP");
    dev_mem_size += nmodes * nnz * sizeof(size_t);

    /* mats_header and lengths */
    size_t sum_mat_length = 0;
    for(size_t m = 0; m < nmodes; ++m) {
        mats_header[m] = mats[m]->values;
        lengths[m] = mats[m]->nrows * stride;
        sum_mat_length += mats[m]->nrows * stride;
    }
    mats_header[nmodes] = mats[nmodes]->values;
    lengths[nmodes] = mats[mode]->nrows * stride;
    sum_mat_length += mats[mode]->nrows * stride;
    /* dev_mats */
    result = sptCudaDuplicateMemoryIndirect(&dev_mats, mats_header, nmodes+1, lengths, hipMemcpyHostToDevice);
    spt_CheckCudaError(result != 0, "CUDA SpTns SpltMTTKRP");
    dev_mem_size += sum_mat_length * sizeof(sptScalar);

    if(nmodes > 4) {
        /* dev_scratch */
        result = hipMalloc((void **) &dev_scratch, nnz * stride * sizeof (sptScalar));
        spt_CheckCudaError(result != 0, "CUDA SpTns MTTKRP");
        result = hipMemset(dev_scratch, 0, nnz * stride * sizeof (sptScalar));
        spt_CheckCudaError(result != 0, "CUDA SpTns MTTKRP");
        dev_mem_size +=  nnz * stride * sizeof (sptScalar);
    }

    sptStopTimer(timer);
    time_h2d = sptElapsedTime(timer);
    gbw_h2d = dev_mem_size / time_h2d /1e9;
    sptPrintElapsedTime(timer, "CUDA SpTns MTTKRP H2D");
    printf("[Bandwidth H2D]: %lf GBytes/sec\n", gbw_h2d);


    // size_t max_nthreads_per_block = 512;    // old run
    size_t max_nthreads_per_block = 256;
    size_t max_nblocks = 32768;
    size_t max_nthreadsy = 16;

    size_t nthreadsx = 0;
    size_t nthreadsy = 0;
    size_t all_nblocks = 0;
    switch(impl_num) {
    case 1: // Naive, 1D
        nthreadsx = 256;
        all_nblocks = (nnz + nthreadsx -1) / nthreadsx;
        break;
    case 2: // 2D
        nthreadsy = R;
        nthreadsx = max_nthreads_per_block / nthreadsy;
        all_nblocks = (nnz + nthreadsx -1) / nthreadsx;
        break;
    case 3: // 2D, rank split
        if(R <= max_nthreadsy)
            nthreadsy = R;
        else
            nthreadsy = max_nthreadsy;
        nthreadsx = max_nthreads_per_block / nthreadsy;
        all_nblocks = (nnz + nthreadsx -1) / nthreadsx;
        break;
    case 4: // 2D, exchange x and y
        nthreadsx = R;
        nthreadsy = max_nthreads_per_block / nthreadsx;
        all_nblocks = (nnz + nthreadsy -1) / nthreadsy;
        break;
    case 5: // 2D, exchange x and y, rank split. Best performance
        if(R <= max_nthreadsy)
            nthreadsx = R;
        else
            nthreadsx = max_nthreadsy;
        nthreadsy = max_nthreads_per_block / nthreadsx;
        all_nblocks = (nnz + nthreadsy -1) / nthreadsy;
        break;
    }
    dim3 dimBlock(nthreadsx, nthreadsy);
    printf("all_nblocks: %zu, nthreadsx: %zu, nthreadsy: %zu\n", all_nblocks, nthreadsx, nthreadsy);


  sptStartTimer(timer);

  for(size_t block_offset = 0; block_offset < all_nblocks; block_offset += max_nblocks) {
    printf("block_offset: %zu\n", block_offset);
    size_t nblocks = (all_nblocks >= block_offset) ? all_nblocks - block_offset: 0;
    if(nblocks > max_nblocks) {
        nblocks = max_nblocks;
    }


    switch(nmodes) {
    case 3:
        switch(impl_num) {
        case 1: // Naive
            printf("Execute spt_MTTKRPKernelNnz3D (%zu, %zu)\n", nblocks, nthreadsx);
            spt_MTTKRPKernelNnz3D<<<nblocks, nthreadsx>>>(
                mode,
                nmodes,
                nnz,
                R,
                stride,
                dev_Xndims,
                dev_Xinds,
                dev_Xvals,
                dev_mats_order,
                dev_mats,
                block_offset);
            break;
        case 2:
            printf("Execute spt_MTTKRPKernelNnzRank3D (%zu, (%u, %u))\n", nblocks, dimBlock.x, dimBlock.y);
            spt_MTTKRPKernelNnzRank3D<<<nblocks, dimBlock>>>(
                mode,
                nmodes,
                nnz,
                R,
                stride,
                dev_Xndims,
                dev_Xinds,
                dev_Xvals,
                dev_mats_order,
                dev_mats,
                block_offset);
            break;
        case 3:
            printf("Execute spt_MTTKRPKernelNnzRankSplit3D (%zu, (%u, %u))\n", nblocks, dimBlock.x, dimBlock.y);
            spt_MTTKRPKernelNnzRankSplit3D<<<nblocks, dimBlock>>>(
                mode,
                nmodes,
                nnz,
                R,
                stride,
                dev_Xndims,
                dev_Xinds,
                dev_Xvals,
                dev_mats_order,
                dev_mats,
                block_offset);
            break;
        case 4:
            printf("Execute spt_MTTKRPKernelRankNnz3D (%zu, (%u, %u))\n", nblocks, dimBlock.x, dimBlock.y);
            spt_MTTKRPKernelRankNnz3D<<<nblocks, dimBlock>>>(
                mode,
                nmodes,
                nnz,
                R,
                stride,
                dev_Xndims,
                dev_Xinds,
                dev_Xvals,
                dev_mats_order,
                dev_mats,
                block_offset);
            break;
        case 5:
            printf("Execute spt_MTTKRPKernelRankSplitNnz3D (%zu, (%u, %u))\n", nblocks, dimBlock.x, dimBlock.y);
            spt_MTTKRPKernelRankSplitNnz3D<<<nblocks, dimBlock>>>(
                mode,
                nmodes,
                nnz,
                R,
                stride,
                dev_Xndims,
                dev_Xinds,
                dev_Xvals,
                dev_mats_order,
                dev_mats,
                block_offset);
            break;
        }   // End switch impl_num
        break;

    case 4: 
        switch(impl_num) {
        default:
            printf("Not support: Execute spt_MTTKRPKernelScratch (%zu, %zu)\n", nblocks, nthreadsx);
            // spt_MTTKRPKernelScratch<<<nblocks, nthreadsx>>>(
            //     mode,
            //     nmodes,
            //     nnz,
            //     R,
            //     stride,
            //     dev_Xndims,
            //     dev_Xinds,
            //     dev_Xvals,
            //     dev_mats_order,
            //     dev_mats,
            //     dev_scratch,
            //     block_offset);
        }   // End switch impl_num
        break;

    default:
        printf("Execute spt_MTTKRPKernelScratch (%zu, %zu)\n", nblocks, nthreadsx);
        spt_MTTKRPKernelScratch<<<nblocks, nthreadsx>>>(
            mode,
            nmodes,
            nnz,
            R,
            stride,
            dev_Xndims,
            dev_Xinds,
            dev_Xvals,
            dev_mats_order,
            dev_mats,
            dev_scratch,
            block_offset);
    }   // End switch nmodes
    result = hipDeviceSynchronize();
    spt_CheckCudaError(result != 0, "CUDA SpTns MTTKRP");

  } // End loop block_offset


    sptStopTimer(timer);
    time_exe = sptElapsedTime(timer);
    gflops_exe = dev_flops / time_exe / 1e9;
    sptPrintElapsedTime(timer, "CUDA SpTns MTTKRP");
    printf("[GFLOPS]: %lf GFlops \n", gflops_exe);

    sptStartTimer(timer);

    dev_mem_size = 0;
    /* Copy back the pointer to dev_mats[nmodes] to the result */
    result = hipMemcpy(&dev_part_prod, dev_mats + nmodes, sizeof dev_part_prod, hipMemcpyDeviceToHost);
    spt_CheckCudaError(result != 0, "CUDA SpTns SpltMTTKRP");
    dev_mem_size += sizeof dev_part_prod;

    result = hipMemcpy(mats[nmodes]->values, dev_part_prod, mats[mode]->nrows * stride * sizeof (sptScalar), hipMemcpyDeviceToHost);
    spt_CheckCudaError(result != 0, "CUDA SpTns SpltMTTKRP");
    dev_mem_size += mats[mode]->nrows * stride * sizeof (sptScalar);

    sptStopTimer(timer);
    time_d2h = sptElapsedTime(timer);
    gbw_d2h = dev_mem_size / time_d2h /1e9;
    sptPrintElapsedTime(timer, "CUDA SpTns MTTKRP D2H");
    printf("[Bandwidth D2H]: %lf GBytes/sec\n", gbw_d2h);
    sptFreeTimer(timer);

    result = hipFree(dev_mats_order);
    spt_CheckCudaError(result != 0, "CUDA SpTns MTTKRP");
    result = hipFree(dev_Xndims);
    spt_CheckCudaError(result != 0, "CUDA SpTns MTTKRP");
    result = hipFree(dev_Xvals);
    spt_CheckCudaError(result != 0, "CUDA SpTns MTTKRP");
    result = hipFree(dev_Xinds);
    spt_CheckCudaError(result != 0, "CUDA SpTns MTTKRP");
    result = hipFree(dev_mats);
    spt_CheckCudaError(result != 0, "CUDA SpTns MTTKRP");
    if(nmodes > 4) {
        result = hipFree(dev_scratch);
        spt_CheckCudaError(result != 0, "CUDA SpTns MTTKRP");
    }
    delete[] Xinds_header;
    delete[] mats_header;
    delete[] lengths;

  return 0;
}



int sptCudaMTTKRPDevice(
    const size_t mode,
    const size_t nmodes,
    const size_t nnz,
    const size_t rank,
    const size_t stride,
    const size_t * Xndims,
    size_t ** const Xinds,
    const sptScalar * Xvals,
    const size_t * dev_mats_order,
    sptScalar ** dev_mats,
    sptScalar * dev_scratch)
{
  int result;

  result = hipMemset(dev_scratch, 0, nnz * rank * sizeof (sptScalar));
  spt_CheckCudaError(result != 0, "CUDA SpTns MTTKRP");

  size_t nthreads = 128;
  const size_t max_nblocks = 32768;
  size_t all_nblocks = (nnz + nthreads -1) / nthreads;

  // sptTimer timer;
  // sptNewTimer(&timer, 0);
  // sptStartTimer(timer);


  for(size_t block_offset = 0; block_offset < all_nblocks; block_offset += max_nblocks) {
    size_t nblocks = all_nblocks - block_offset;
    if(nblocks > max_nblocks) {
        nblocks = max_nblocks;
    }
    spt_MTTKRPKernelScratch<<<nblocks, nthreads>>>(
        mode,
        nmodes,
        nnz,
        rank,
        stride,
        Xndims,
        Xinds,
        Xvals,
        dev_mats_order,
        dev_mats,
        dev_scratch,
        block_offset
        );
    result = hipDeviceSynchronize();
    spt_CheckCudaError(result != 0, "CUDA SpTns MTTKRP");
  }

  // sptStopTimer(timer);
  // sptPrintElapsedTime(timer, "CUDA SpTns MTTKRP");
  // sptFreeTimer(timer);


  return 0;
}
