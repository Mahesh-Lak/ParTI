/*
    This file is part of ParTI!.

    ParTI! is free software: you can redistribute it and/or modify
    it under the terms of the GNU Lesser General Public License as
    published by the Free Software Foundation, either version 3 of
    the License, or (at your option) any later version.

    ParTI! is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU Lesser General Public
    License along with ParTI!.
    If not, see <http://www.gnu.org/licenses/>.
*/

#include <ParTI.h>
#include "sptensor.h"
#include "../cudawrap.h"
#include "mttkrp_cuda_kernels.h"



/**
 * CUDA parallelized Matriced sparse tensor times a sequence of dense matrix Khatri-Rao products (MTTKRP) on a specified mode
 * @param[out] mats[nmodes]    the result of MTTKRP, a dense matrix, with size
 * ndims[mode] * R
 * @param[in]  X    the sparse tensor input X
 * @param[in]  mats    (N+1) dense matrices, with mats[nmodes] as temporary
 * @param[in]  mats_order    the order of the Khatri-Rao products
 * @param[in]  mode   the mode on which the MTTKRP is performed
 * @param[in]  scratch an temporary array to store intermediate results, space assigned before this function
 *
 * This function uses support arbitrary-order sparse tensors with Khatri-Rao
 * products of dense factor matrices, the output is the updated dense matrix for the "mode".
 * In this version, atomic function to lock the global reduction and a large
 * scratch is used to maximize parallelism. (To be optimized)
 */
int sptCudaMTTKRPOneKernel(
    sptSparseTensor const * const X,
    sptMatrix ** const mats,     // mats[nmodes] as temporary space.
    size_t * const mats_order,    // Correspond to the mode order of X.
    size_t const mode,
    size_t const impl_num) 
{
    size_t const nmodes = X->nmodes;
    size_t const nnz = X->nnz;
    size_t const * const ndims = X->ndims;
    size_t const R = mats[mode]->ncols;
    size_t const stride = mats[mode]->stride;
    int result;

    double time_h2d, time_exe, time_d2h;
    double gbw_h2d, gflops_exe, gbw_d2h;
    sptTimer timer;
    sptNewTimer(&timer, 0);

    /* Check the mats. */
    for(size_t i=0; i<nmodes; ++i) {
        if(mats[i]->ncols != mats[nmodes]->ncols) {
            spt_CheckError(SPTERR_SHAPE_MISMATCH, "CUDA SpTns MTTKRP", "mats[i]->cols != mats[nmodes]->ncols");
        }
        if(mats[i]->nrows != ndims[i]) {
            spt_CheckError(SPTERR_SHAPE_MISMATCH, "CUDA SpTns MTTKRP", "mats[i]->nrows != ndims[i]");
        }
    }


    /* Transfer tensor and matrices */
    /* dev_mats_order: 1st gpu. */
    size_t * dev_mats_order;
    /* dev_Xndims: 1st gpu. */
    size_t * dev_Xndims;
    /* dev_Xvals: 1st gpu. */
    sptScalar * dev_Xvals;
    /* Xinds_header: 1st cpu, 2nd cpu (ghost pointers) */
    size_t ** Xinds_header = new size_t *[nmodes];
    /* dev_Xinds: 1st gpu, 2nd gpu. */
    size_t ** dev_Xinds;
    /* mats_header: 1st cpu, 2nd cpu (ghost pointers) */
    sptScalar ** mats_header = new sptScalar *[nmodes+1];
    /* lengths: 1st cpu, store the lengths of mats */
    size_t * const lengths = new size_t[nmodes+1];
    /* dev_mats: 1st gpu, 2nd gpu. */
    sptScalar ** dev_mats;
    /* dev_scratch: 1st gpu. */
    sptScalar * dev_scratch;
    /* the pointer to dev_mats[nmodes] */
    sptScalar *dev_part_prod;  
    size_t dev_mem_size = 0;
    size_t dev_flops = 2 * nnz * R + (nmodes - 1) * R;


    sptStartTimer(timer);

    /* dev_mats_order */
    result = sptCudaDuplicateMemory(&dev_mats_order, mats_order, nmodes * sizeof (size_t), hipMemcpyHostToDevice);
    spt_CheckCudaError(result != 0, "CUDA SpTns SpltMTTKRP");
    dev_mem_size += nmodes * sizeof (size_t);

    /* dev_Xndims */
    result = sptCudaDuplicateMemory(&dev_Xndims, ndims, nmodes * sizeof (size_t), hipMemcpyHostToDevice);
    spt_CheckCudaError(result != 0, "CUDA SpTns SpltMTTKRP");
    dev_mem_size += nmodes * sizeof (size_t);

    /* dev_Xvals */
    result = sptCudaDuplicateMemory(&dev_Xvals, X->values.data, nnz * sizeof (sptScalar), hipMemcpyHostToDevice);
    spt_CheckCudaError(result != 0, "CUDA SpTns SpltMTTKRP");
    dev_mem_size += nnz * sizeof (sptScalar);

    /* Xinds_header */
    for(size_t m = 0; m < nmodes; ++m) {
        Xinds_header[m] = X->inds[m].data;
    }
    /* dev_Xinds */
    result = sptCudaDuplicateMemoryIndirect(&dev_Xinds, Xinds_header, nmodes, nnz, hipMemcpyHostToDevice);
    spt_CheckCudaError(result != 0, "CUDA SpTns SpltMTTKRP");
    dev_mem_size += nmodes * nnz * sizeof(size_t);

    /* mats_header and lengths */
    size_t sum_mat_length = 0;
    for(size_t m = 0; m < nmodes; ++m) {
        mats_header[m] = mats[m]->values;
        lengths[m] = mats[m]->nrows * stride;
        sum_mat_length += mats[m]->nrows * stride;
    }
    mats_header[nmodes] = mats[nmodes]->values;
    lengths[nmodes] = mats[mode]->nrows * stride;
    sum_mat_length += mats[mode]->nrows * stride;
    /* dev_mats */
    result = sptCudaDuplicateMemoryIndirect(&dev_mats, mats_header, nmodes+1, lengths, hipMemcpyHostToDevice);
    spt_CheckCudaError(result != 0, "CUDA SpTns SpltMTTKRP");
    dev_mem_size += sum_mat_length * sizeof(sptScalar);

    if(nmodes > 4) {
        /* dev_scratch */
        result = hipMalloc((void **) &dev_scratch, nnz * stride * sizeof (sptScalar));
        spt_CheckCudaError(result != 0, "CUDA SpTns MTTKRP");
        result = hipMemset(dev_scratch, 0, nnz * stride * sizeof (sptScalar));
        spt_CheckCudaError(result != 0, "CUDA SpTns MTTKRP");
        dev_mem_size +=  nnz * stride * sizeof (sptScalar);
    }

    sptStopTimer(timer);
    time_h2d = sptElapsedTime(timer);
    gbw_h2d = dev_mem_size / time_h2d /1e9;
    sptPrintElapsedTime(timer, "CUDA SpTns MTTKRP H2D");
    printf("[Bandwidth H2D]: %lf GBytes/sec\n", gbw_h2d);


    // size_t max_nthreads_per_block = 512;    // old run
    size_t max_nthreads_per_block = 256;
    size_t max_nblocks = 32768;
    size_t max_nthreadsy = 16;

    size_t nthreadsx = 0;
    size_t nthreadsy = 0;
    size_t all_nblocks = 0;
    size_t nblocks = 0;
    switch(impl_num) {
    // case 1:
    case 11: // Naive, 1D
        if(nnz < max_nthreads_per_block) {
            nthreadsx = nnz;
            nblocks = 1;
        } else {
            nthreadsx = max_nthreads_per_block;
            all_nblocks = (nnz + nthreadsx -1) / nthreadsx;
            if(all_nblocks < max_nblocks) {
                nblocks = all_nblocks;
            } else {
                nblocks = max_nblocks;
            }
        }
        break;
    // case 2: // 2D
    case 12:
        if(R <= max_nthreadsy)
            nthreadsy = R;
        else
            nthreadsy = max_nthreadsy;
        nthreadsx = max_nthreads_per_block / nthreadsy;

        if(nnz < nthreadsx) {
            nthreadsx = nnz;
            nblocks = 1;
        } else {
            all_nblocks = (nnz + nthreadsx -1) / nthreadsx;
            if(all_nblocks < max_nblocks) {
                nblocks = all_nblocks;
            } else {
                nblocks = max_nblocks;
            }   
        }
        break;
    // case 3: // 2D, rank split
    //     if(R <= max_nthreadsy)
    //         nthreadsy = R;
    //     else
    //         nthreadsy = max_nthreadsy;
    //     nthreadsx = max_nthreads_per_block / nthreadsy;
    //     all_nblocks = (nnz + nthreadsx -1) / nthreadsx;
    //     break;
    // case 4: // 2D, exchange x and y
    //     nthreadsx = R;
    //     nthreadsy = max_nthreads_per_block / nthreadsx;
    //     all_nblocks = (nnz + nthreadsy -1) / nthreadsy;
    //     break;
    // case 5:
    case 15: // 2D, exchange x and y, rank split. Best performance
    case 16:
        if(R <= max_nthreadsy)
            nthreadsx = R;
        else
            nthreadsx = max_nthreadsy;
        nthreadsy = max_nthreads_per_block / nthreadsx;

        if(nnz < nthreadsy) {
            nthreadsy = nnz;
            nblocks = 1;
        } else {
            all_nblocks = (nnz + nthreadsy -1) / nthreadsy;
            if(all_nblocks < max_nblocks) {
                nblocks = all_nblocks;
            } else {
                nblocks = max_nblocks;
            }   
        }
        break;
    }
    dim3 dimBlock(nthreadsx, nthreadsy);
    printf("all_nblocks: %zu, nthreadsx: %zu, nthreadsy: %zu\n", all_nblocks, nthreadsx, nthreadsy);


    sptStartTimer(timer);

    switch(nmodes) {
    case 3:
        switch(impl_num) {
        // case 1:
        case 11: // Naive
            printf("Execute spt_MTTKRPKernelNnz3DOneKernel (%zu, %zu)\n", nblocks, nthreadsx);
            spt_MTTKRPKernelNnz3DOneKernel<<<nblocks, nthreadsx>>>(
                mode,
                nmodes,
                nnz,
                R,
                stride,
                dev_Xndims,
                dev_Xinds,
                dev_Xvals,
                dev_mats_order,
                dev_mats);
            break;
        // case 2:
        case 12:
            printf("Execute spt_MTTKRPKernelRankNnz3DOneKernel (%zu, (%u, %u))\n", nblocks, dimBlock.x, dimBlock.y);
            spt_MTTKRPKernelRankNnz3DOneKernel<<<nblocks, dimBlock>>>(
                mode,
                nmodes,
                nnz,
                R,
                stride,
                dev_Xndims,
                dev_Xinds,
                dev_Xvals,
                dev_mats_order,
                dev_mats);
            break;
        case 3:
            printf("Execute spt_MTTKRPKernelNnzRankSplit3D (%zu, (%u, %u))\n", nblocks, dimBlock.x, dimBlock.y);
            // spt_MTTKRPKernelNnzRankSplit3D<<<nblocks, dimBlock>>>(
            //     mode,
            //     nmodes,
            //     nnz,
            //     R,
            //     stride,
            //     dev_Xndims,
            //     dev_Xinds,
            //     dev_Xvals,
            //     dev_mats_order,
            //     dev_mats,
            //     block_offset);
            break;
        case 4:
            printf("Execute spt_MTTKRPKernelRankNnz3D (%zu, (%u, %u))\n", nblocks, dimBlock.x, dimBlock.y);
            // spt_MTTKRPKernelRankNnz3D<<<nblocks, dimBlock>>>(
            //     mode,
            //     nmodes,
            //     nnz,
            //     R,
            //     stride,
            //     dev_Xndims,
            //     dev_Xinds,
            //     dev_Xvals,
            //     dev_mats_order,
            //     dev_mats,
            //     block_offset);
            break;
        // case 5:
        case 15:
            printf("Execute spt_MTTKRPKernelRankSplitNnz3DOneKernel (%zu, (%u, %u))\n", nblocks, dimBlock.x, dimBlock.y);
            spt_MTTKRPKernelRankSplitNnz3DOneKernel<<<nblocks, dimBlock>>>(
                mode,
                nmodes,
                nnz,
                R,
                stride,
                dev_Xndims,
                dev_Xinds,
                dev_Xvals,
                dev_mats_order,
                dev_mats);
            break;
        case 16:
            printf("Execute spt_MTTKRPKernelRankSplitNnzRB3DOneKernel (%zu, (%u, %u))\n", nblocks, dimBlock.x, dimBlock.y);
            spt_MTTKRPKernelRankSplitNnzRB3DOneKernel<<<nblocks, dimBlock>>>(
                mode,
                nmodes,
                nnz,
                R,
                stride,
                dev_Xndims,
                dev_Xinds,
                dev_Xvals,
                dev_mats_order,
                dev_mats);
            break;
        }   // End switch impl_num
        break;

    case 4: 
        switch(impl_num) {
        default:
            printf("Not support: Execute spt_MTTKRPKernelScratch (%zu, %zu)\n", nblocks, nthreadsx);
            // spt_MTTKRPKernelScratch<<<nblocks, nthreadsx>>>(
            //     mode,
            //     nmodes,
            //     nnz,
            //     R,
            //     stride,
            //     dev_Xndims,
            //     dev_Xinds,
            //     dev_Xvals,
            //     dev_mats_order,
            //     dev_mats,
            //     dev_scratch,
            //     block_offset);
        }   // End switch impl_num
        break;

    default:
        printf("Execute spt_MTTKRPKernelScratch (%zu, %zu)\n", nblocks, nthreadsx);
        // spt_MTTKRPKernelScratch<<<nblocks, nthreadsx>>>(
        //     mode,
        //     nmodes,
        //     nnz,
        //     R,
        //     stride,
        //     dev_Xndims,
        //     dev_Xinds,
        //     dev_Xvals,
        //     dev_mats_order,
        //     dev_mats,
        //     dev_scratch,
        //     block_offset);
    }   // End switch nmodes
    result = hipDeviceSynchronize();
    spt_CheckCudaError(result != 0, "CUDA SpTns MTTKRP");



    sptStopTimer(timer);
    time_exe = sptElapsedTime(timer);
    gflops_exe = dev_flops / time_exe / 1e9;
    sptPrintElapsedTime(timer, "CUDA SpTns MTTKRP");
    printf("[GFLOPS]: %lf GFlops \n", gflops_exe);

    sptStartTimer(timer);

    dev_mem_size = 0;
    /* Copy back the pointer to dev_mats[nmodes] to the result */
    result = hipMemcpy(&dev_part_prod, dev_mats + nmodes, sizeof dev_part_prod, hipMemcpyDeviceToHost);
    spt_CheckCudaError(result != 0, "CUDA SpTns SpltMTTKRP");
    dev_mem_size += sizeof dev_part_prod;

    result = hipMemcpy(mats[nmodes]->values, dev_part_prod, mats[mode]->nrows * stride * sizeof (sptScalar), hipMemcpyDeviceToHost);
    spt_CheckCudaError(result != 0, "CUDA SpTns SpltMTTKRP");
    dev_mem_size += mats[mode]->nrows * stride * sizeof (sptScalar);

    sptStopTimer(timer);
    time_d2h = sptElapsedTime(timer);
    gbw_d2h = dev_mem_size / time_d2h /1e9;
    sptPrintElapsedTime(timer, "CUDA SpTns MTTKRP D2H");
    printf("[Bandwidth D2H]: %lf GBytes/sec\n", gbw_d2h);
    sptFreeTimer(timer);

    result = hipFree(dev_mats_order);
    spt_CheckCudaError(result != 0, "CUDA SpTns MTTKRP");
    result = hipFree(dev_Xndims);
    spt_CheckCudaError(result != 0, "CUDA SpTns MTTKRP");
    result = hipFree(dev_Xvals);
    spt_CheckCudaError(result != 0, "CUDA SpTns MTTKRP");
    result = hipFree(dev_Xinds);
    spt_CheckCudaError(result != 0, "CUDA SpTns MTTKRP");
    result = hipFree(dev_mats);
    spt_CheckCudaError(result != 0, "CUDA SpTns MTTKRP");
    if(nmodes > 4) {
        result = hipFree(dev_scratch);
        spt_CheckCudaError(result != 0, "CUDA SpTns MTTKRP");
    }
    delete[] Xinds_header;
    delete[] mats_header;
    delete[] lengths;

  return 0;
}


