#include "hip/hip_runtime.h"
#include <SpTOL.h>
#include "sptensor.h"

__global__ static void spt_DotMulKernel(size_t nnz, sptScalar *Z_val, sptScalar *X_val, sptScalar *Y_val) 
{
    const size_t tidx = threadIdx.x;
    const size_t i = blockIdx.x * blockDim.x + tidx;

    if(i < nnz) {
        Z_val[i] = X_val[i] * Y_val[i];
    }
    __syncthreads();
}



/**
 * Element wise multiply two sparse tensors
 * @param[out] Z the result of X*Y, should be uninitialized
 * @param[in]  X the input X
 * @param[in]  Y the input Y
 */
int sptCudaSparseTensorDotMul(sptSparseTensor *Z, const sptSparseTensor *X, const sptSparseTensor *Y) {
    size_t i;
    int result;
    /* Ensure X and Y are in same shape */
    if(Y->nmodes != X->nmodes) {
        spt_CheckError(SPTERR_SHAPE_MISMATCH, "CUDA SpTns DotMul", "shape mismatch");
    }
    for(i = 0; i < X->nmodes; ++i) {
        if(Y->ndims[i] != X->ndims[i]) {
            spt_CheckError(SPTERR_SHAPE_MISMATCH, "CUDA SpTns DotMul", "shape mismatch");
        }
    }
    /* Ensure X and Y have exactly the same nonzero distribution */
    if(Y->nnz != X->nnz) {
        spt_CheckError(SPTERR_SHAPE_MISMATCH, "SpTns DotMul", "nonzero distribution mismatch");
    }
    size_t nnz = X->nnz;

    sptCopySparseTensor(Z, X);

    sptScalar *X_val = NULL;
    result = hipMalloc((void **) &X_val, X->nnz * sizeof (sptScalar));
    spt_CheckCudaError(result != 0, "CUDA SpTns DotMul");
    result = hipMemcpy(X_val, X->values.data, X->nnz * sizeof (sptScalar), hipMemcpyHostToDevice);
    spt_CheckCudaError(result != 0, "CUDA SpTns DotMul");

    sptScalar *Y_val = NULL;
    result = hipMalloc((void **) &Y_val, Y->nnz * sizeof (sptScalar));
    spt_CheckCudaError(result != 0, "CUDA SpTns DotMul");
    result = hipMemcpy(Y_val, Y->values.data, Y->nnz * sizeof (sptScalar), hipMemcpyHostToDevice);
    spt_CheckCudaError(result != 0, "CUDA SpTns DotMul");

    sptScalar *Z_val = NULL;
    result = hipMalloc((void **) &Z_val, X->nnz * sizeof (sptScalar));
    spt_CheckCudaError(result != 0, "CUDA SpTns DotMul");
    result = hipMemset(Z_val, 0, X->nnz * sizeof (sptScalar));
    spt_CheckCudaError(result != 0, "CUDA SpTns DotMul");

    size_t nthreads = 128;
    size_t nblocks = (nnz + nthreads -1)/ nthreads;

    sptTimer timer;
    sptNewTimer(&timer, 0);
    sptStartTimer(timer);

    spt_DotMulKernel<<<nblocks, nthreads>>>(nnz, Z_val, X_val, Y_val);
    result = hipDeviceSynchronize();

    sptStopTimer(timer);
    sptPrintElapsedTime(timer, "CUDA  SpTns DotMul");
    sptFreeTimer(timer);

    hipMemcpy(Z->values.data, Z_val, Z->nnz * sizeof (sptScalar), hipMemcpyDeviceToHost);

    result = hipFree(X_val);
    spt_CheckCudaError(result != 0, "CUDA SpTns DotMul");
    result = hipFree(Y_val);
    spt_CheckCudaError(result != 0, "CUDA SpTns DotMul");
    result = hipFree(Z_val);
    spt_CheckCudaError(result != 0, "CUDA SpTns DotMul");

    /* Check whether elements become zero after adding.
       If so, fill the gap with the [nnz-1]'th element.
    */
    spt_SparseTensorCollectZeros(Z);
    /* Sort the indices */
    sptSparseTensorSortIndex(Z);
    return 0;
}
