#include "hip/hip_runtime.h"
/*
    This file is part of ParTI!.

    ParTI! is free software: you can redistribute it and/or modify
    it under the terms of the GNU Lesser General Public License as
    published by the Free Software Foundation, either version 3 of
    the License, or (at your option) any later version.

    ParTI! is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU Lesser General Public
    License along with ParTI!.
    If not, see <http://www.gnu.org/licenses/>.
*/

#include <ParTI.h>
#include "sptensor.h"
#include <hip/hip_runtime.h>
#include "../cudawrap.h"


/* A thread block compute a sub-tensor */
__global__ static void spt_MTTKRPKernelSM(
  const size_t mode,
  const size_t nmodes,
  const size_t nnz,
  const size_t R,
  const size_t stride,
  const size_t * Xndims,
  const size_t nsplits,
  size_t ** const split_Xndims,
  size_t * const split_nnz,
  size_t *** const split_Xinds,
  sptScalar ** const split_Xvals,
  const size_t * dev_mats_order,
  sptScalar ** dev_mats,
  size_t block_offset) 
{
  const size_t tidx = threadIdx.x;
  const size_t bidx = blockIdx.x + block_offset;

#if 1
  extern __shared__ sptScalar sm_pool[];
  size_t const nmats = nmodes - 1;
  size_t * split_low = split_Xndims[bidx];
  size_t const block_nnz = split_nnz[bidx];

  /* data in shared memory */
  size_t * sm_dev_mats_order = (size_t *)sm_pool; // dev_mats_order
  size_t * sm_split_low = sm_dev_mats_order + nmats; // split_low
  size_t * sm_split_high = sm_split_low + nmodes; // split_high

  if(nmats + 2 * nmodes < blockDim.x) {
    if(tidx < nmats) {
      sm_dev_mats_order[tidx] = dev_mats_order[tidx];
    } else if (tidx >= nmats && tidx < nmats + 2 * nmodes) {
      sm_split_low[tidx - nmats] = split_low[tidx - nmats];
    }
  } else {
    if(tidx < nmats) {
      sm_dev_mats_order[tidx] = dev_mats_order[tidx];
    }
    if (tidx < 2 * nmodes) {
      sm_split_low[tidx] = split_low[tidx];
    }
  }
  __syncthreads();

  sptScalar * sm_dev_scratch = (sptScalar *)(sm_split_high + nmodes);  // mode matrix
  sptScalar * sm_times_mat = sm_dev_scratch + block_nnz * R; // Timing matrix, this replace need to be reused.

  size_t times_mat_mode = sm_dev_mats_order[0];
  sptScalar * times_mat = dev_mats[times_mat_mode];
  size_t times_mat_nrows = sm_split_high[times_mat_mode] - sm_split_low[times_mat_mode];
  if(tidx < times_mat_nrows) {
    for(size_t r=0; r<R; ++r)
      sm_times_mat[tidx * stride + r] = times_mat[(tidx+sm_split_low[times_mat_mode]) * stride + r];
  }
  __syncthreads();
  size_t * times_inds = split_Xinds[bidx][times_mat_mode];
  size_t index;
  sptScalar entry;
  if(tidx < block_nnz) {
    index = times_inds[tidx] - sm_split_low[times_mat_mode];
    entry = split_Xvals[bidx][tidx];
    for(size_t r=0; r<R; ++r) {
      sm_dev_scratch[tidx * stride + r] = entry * sm_times_mat[index * stride + r];
    }
  }
  __syncthreads();

  for(size_t i=1; i<nmats; ++i) {
    times_mat_mode = sm_dev_mats_order[i];
    times_mat = dev_mats[times_mat_mode];
    times_mat_nrows = sm_split_high[times_mat_mode] - sm_split_low[times_mat_mode];
    if(tidx < times_mat_nrows) {
      for(size_t r=0; r<R; ++r)
        sm_times_mat[tidx * stride + r] = times_mat[(tidx+sm_split_low[times_mat_mode]) * stride + r];
    }
    __syncthreads();
    times_inds = split_Xinds[bidx][times_mat_mode];
    if(tidx < block_nnz) {
      index = times_inds[tidx] - sm_split_low[times_mat_mode];
      for(size_t r=0; r<R; ++r) {
        sm_dev_scratch[tidx * stride + r] *= sm_times_mat[index * stride + r];
      }
    }
    __syncthreads();
  }

  size_t const * const mode_ind = split_Xinds[bidx][mode];
  sptScalar * const mvals = dev_mats[nmodes];
  size_t mmat_nrows = sm_split_high[mode] - sm_split_low[mode];
  if(tidx < mmat_nrows) {
    for(size_t r=0; r<R; ++r)
      sm_times_mat[tidx * stride + r] = mvals[(tidx+sm_split_low[mode]) * stride + r];
  }
  __syncthreads();

  
  if(tidx < block_nnz) {
    size_t const mode_i = mode_ind[tidx] - sm_split_low[mode];
    for(size_t r=0; r<R; ++r) {
      atomicAdd(&(sm_times_mat[mode_i * stride + r]), sm_dev_scratch[tidx * stride + r]);
    }
  }
  __syncthreads();


  if(tidx < mmat_nrows) {
    for(size_t r=0; r<R; ++r)
      atomicAdd(&(mvals[(tidx+sm_split_low[mode]) * stride + r]), sm_times_mat[tidx * stride + r]);
  }
  __syncthreads();
#endif

}




/**
 * CUDA parallelized Matriced sparse tensor times a sequence of dense matrix Khatri-Rao products (MTTKRP) on a specified mode
 * @param[out] mats[nmodes]    the result of MTTKRP, a dense matrix, with size
 * ndims[mode] * R
 * @param[in]  X    the sparse tensor input X
 * @param[in]  mats    (N+1) dense matrices, with mats[nmodes] as temporary
 * @param[in]  mats_order    the order of the Khatri-Rao products
 * @param[in]  mode   the mode on which the MTTKRP is performed
 * @param[in]  scratch an temporary array to store intermediate results, space assigned before this function
 *
 * This function uses support arbitrary-order sparse tensors with Khatri-Rao
 * products of dense factor matrices, the output is the updated dense matrix for the "mode".
 * In this version, atomic function to lock the global reduction and a large
 * scratch is used to maximize parallelism. (To be optimized)
 */
int sptCudaMTTKRPSM(sptSparseTensor const * const X,
  sptMatrix ** const mats,     // mats[nmodes] as temporary space.
  sptSizeVector const * const mats_order,    // Correspond to the mode order of X.
  size_t const mode) {

  size_t const memory_size = 49152; // Shared memory size
  size_t const nmodes = X->nmodes;
  size_t const nnz = X->nnz;
  size_t const * const ndims = X->ndims;
  size_t const R = mats[mode]->ncols;
  size_t const stride = mats[mode]->stride;
  size_t const nmats = nmodes - 1;
  int result;

  /* Check the mats. */
  for(size_t i=0; i<nmodes; ++i) {
      if(mats[i]->ncols != mats[nmodes]->ncols) {
          spt_CheckError(SPTERR_SHAPE_MISMATCH, "CUDA SpTns MTTKRP", "mats[i]->cols != mats[nmodes]->ncols");
      }
      if(mats[i]->nrows != ndims[i]) {
          spt_CheckError(SPTERR_SHAPE_MISMATCH, "CUDA SpTns MTTKRP", "mats[i]->nrows != ndims[i]");
      }
  }


  size_t const memory_step = (size_t) (0.9 * memory_size/(sizeof(size_t)*R*2));
  printf("memory_step: %lu\n", memory_step);
  size_t *steps = (size_t*)malloc(nmodes * sizeof (size_t));
  for(size_t i=0; i<nmodes; ++i)
    steps[i] = memory_step;
  spt_SplitResult *splits;
  size_t nsplits;
  sptAssert(spt_SparseTensorGetAllSplits(&splits, &nsplits, X, steps, NULL, 1) == 0);
  spt_SparseTensorDumpAllSplits(splits, nsplits, stdout);


  /* Transfer tensor and matrices */
  size_t * Xndims = NULL;
  result = sptCudaDuplicateMemory(&Xndims, ndims, nmodes * sizeof (size_t), hipMemcpyHostToDevice);
  spt_CheckCudaError(result != 0, "CUDA SpTns MTTKRP");

  size_t * dev_mats_order = NULL;
  result = sptCudaDuplicateMemory(&dev_mats_order,  mats_order->data, nmats * sizeof(size_t), hipMemcpyHostToDevice);
  spt_CheckCudaError(result != 0, "CUDA SpTns MTTKRP");

  sptScalar ** tmp_mats = NULL;
  tmp_mats = (sptScalar **)malloc((nmodes+1) * sizeof(sptScalar*));
  for(size_t i=0; i<nmodes+1; ++i) {
    result = sptCudaDuplicateMemory(&(tmp_mats[i]), mats[i]->values, 
      mats[i]->nrows * mats[i]->stride * sizeof(sptScalar), hipMemcpyHostToDevice);
    spt_CheckCudaError(result != 0, "CUDA SpTns MTTKRP");
  }
  result = hipMemset(tmp_mats[nmodes], 0, mats[nmodes]->nrows * mats[nmodes]->stride * sizeof (sptScalar));
  spt_CheckCudaError(result != 0, "CUDA SpTns MTTKRP");
  sptScalar ** dev_mats = NULL;   // array of pointer to device memory
  result = sptCudaDuplicateMemory(&dev_mats, tmp_mats, (nmodes+1) * sizeof (sptScalar*), hipMemcpyHostToDevice);
  spt_CheckCudaError(result != 0, "CUDA SpTns MTTKRP");


  size_t ** tmp_Xndims = NULL;
  tmp_Xndims = (size_t **)malloc(nsplits * sizeof(size_t*));
  for(size_t i=0; i<nsplits; ++i) {
    result = sptCudaDuplicateMemory(&(tmp_Xndims[i]), splits[i].inds_low, 
      2 * nmodes * sizeof(size_t), hipMemcpyHostToDevice);
    spt_CheckCudaError(result != 0, "CUDA SpTns MTTKRP");
  }
  size_t ** split_Xndims = NULL;   // array of pointer to device memory
  result = hipMalloc((void***)&split_Xndims, nsplits * sizeof(size_t*));
  spt_CheckCudaError(result != 0, "CUDA SpTns MTTKRP");
  result = hipMemcpy(split_Xndims, tmp_Xndims, nsplits * sizeof (size_t*), hipMemcpyHostToDevice);
  spt_CheckCudaError(result != 0, "CUDA SpTns MTTKRP");


  size_t * tmp_nnz = (size_t *)malloc(nsplits * sizeof(size_t));
  for(size_t i=0; i<nsplits; ++i) {
    tmp_nnz[i] = splits[i].tensor.nnz;
  }
  size_t * split_nnz = NULL;
  result = sptCudaDuplicateMemory(&split_nnz, tmp_nnz, nsplits * sizeof(size_t), hipMemcpyHostToDevice);
  spt_CheckCudaError(result != 0, "CUDA SpTns MTTKRP");
  free(tmp_nnz);


  size_t *** tmp1_split_Xinds = (size_t ***)malloc(nsplits * sizeof(size_t**));
  for(size_t i=0; i<nsplits; ++i) {
    tmp1_split_Xinds[i] = (size_t **)malloc(nmodes * sizeof(size_t*));
    for(size_t m=0; m<nmodes; ++m) {
      result = sptCudaDuplicateMemory(&(tmp1_split_Xinds[i][m]), splits[i].tensor.inds[m].data, 
        splits[i].tensor.nnz * sizeof(size_t), hipMemcpyHostToDevice);
      spt_CheckCudaError(result != 0, "CUDA SpTns MTTKRP");
    }  
  }
  size_t *** tmp2_split_Xinds = (size_t ***)malloc(nsplits * sizeof(size_t**));
  for(size_t i=0; i<nsplits; ++i) {
    result = hipMalloc((void***)&(tmp2_split_Xinds[i]), nmodes * sizeof(size_t*));
    spt_CheckCudaError(result != 0, "CUDA SpTns MTTKRP");
    result = hipMemcpy(tmp2_split_Xinds[i], tmp1_split_Xinds[i], nmodes * sizeof(size_t*), hipMemcpyHostToDevice);
    spt_CheckCudaError(result != 0, "CUDA SpTns MTTKRP");
  }
  size_t *** split_Xinds = NULL;   // array of pointer to device memory
  result = hipMalloc((void ****) &split_Xinds, nsplits * sizeof(size_t**));
  spt_CheckCudaError(result != 0, "CUDA SpTns MTTKRP");
  result = hipMemcpy(split_Xinds, tmp2_split_Xinds, nsplits * sizeof (size_t**), hipMemcpyHostToDevice);
  spt_CheckCudaError(result != 0, "CUDA SpTns MTTKRP");


  sptScalar ** split_Xvals = NULL;
  sptScalar ** tmp_split_Xvals = (sptScalar **)malloc(nsplits * sizeof(sptScalar*));
  for(size_t i=0; i<nsplits; ++i) {
    result = sptCudaDuplicateMemory(&(tmp_split_Xvals[i]), splits[i].tensor.values.data, 
      splits[i].tensor.nnz * sizeof (sptScalar), hipMemcpyHostToDevice);
    spt_CheckCudaError(result != 0, "CUDA SpTns MTTKRP");
  }
  result = hipMalloc((void ***) &split_Xvals, nsplits * sizeof(sptScalar*));
  spt_CheckCudaError(result != 0, "CUDA SpTns MTTKRP");
  result = hipMemcpy(split_Xvals, tmp_split_Xvals, nsplits * sizeof (sptScalar*), hipMemcpyHostToDevice);
  spt_CheckCudaError(result != 0, "CUDA SpTns MTTKRP");


  const size_t nthreads = 128;
  const size_t max_nblocks = 32768;
  printf("nsplits: %lu, nthreads: %lu\n", nsplits, nthreads);
  size_t max_block_nnz = 0;
  for(size_t i=0; i<nsplits; ++i) {
    if(max_block_nnz < splits[i].tensor.nnz)
      max_block_nnz = splits[i].tensor.nnz;
  }
  size_t allocate_sm_size = (nmats + 2 * nmodes) * sizeof(size_t) + 2 * max_block_nnz * R * sizeof(sptScalar);
  printf("max_block_nnz: %lu\n", max_block_nnz);
  printf("allocate_sm_size: %lu, given shared memory size: %lu\n", allocate_sm_size, memory_size);
  sptAssert (allocate_sm_size < memory_size);


  sptTimer timer;
  sptNewTimer(&timer, 0);
  sptStartTimer(timer);

  for(size_t block_offset = 0; block_offset < nsplits; block_offset += max_nblocks) {
    size_t nblocks = nsplits - block_offset;
    if(nblocks > max_nblocks) {
        nblocks = max_nblocks;
    }
    spt_MTTKRPKernelSM<<<nblocks, nthreads, memory_size>>>(
        mode,
        nmodes,
        nnz,
        R,
        stride,
        Xndims,
        nsplits,
        split_Xndims,
        split_nnz,
        split_Xinds,
        split_Xvals,
        dev_mats_order,
        dev_mats,
        block_offset
        );
    result = hipDeviceSynchronize();
    spt_CheckCudaError(result != 0, "CUDA SpTns MTTKRP");
  }


  sptStopTimer(timer);
  sptPrintElapsedTime(timer, "CUDA SpTns MTTKRP");
  sptFreeTimer(timer);


  result = hipMemcpy(mats[nmodes]->values, tmp_mats[nmodes], mats[nmodes]->nrows * mats[nmodes]->stride * sizeof (sptScalar), hipMemcpyDeviceToHost);
  spt_CheckCudaError(result != 0, "CUDA SpTns MTTKRP copy back");


  result = hipFree(Xndims);
  spt_CheckCudaError(result != 0, "CUDA SpTns MTTKRP");
  result = hipFree(dev_mats_order);
  spt_CheckCudaError(result != 0, "CUDA SpTns MTTKRP");
  for(size_t i=0; i<nsplits; ++i) {
    result = hipFree(tmp_split_Xvals[i]);
    spt_CheckCudaError(result != 0, "CUDA SpTns MTTKRP");
  }
  free(tmp_split_Xvals);
  result = hipFree(split_Xvals);
  spt_CheckCudaError(result != 0, "CUDA SpTns MTTKRP");
  
  for(size_t i=0; i<nsplits; ++i) {
    for(size_t m=0; m<nmodes; ++m) {
      result = hipFree(tmp1_split_Xinds[i][m]);
      spt_CheckCudaError(result != 0, "CUDA SpTns MTTKRP");
    }
    free(tmp1_split_Xinds[i]);
  }
  free(tmp1_split_Xinds);
  for(size_t i=0; i<nsplits; ++i) {
    result = hipFree(tmp2_split_Xinds[i]);
    spt_CheckCudaError(result != 0, "CUDA SpTns MTTKRP");
  }
  free(tmp2_split_Xinds);
  result = hipFree(split_Xinds);
  spt_CheckCudaError(result != 0, "CUDA SpTns MTTKRP");

  for(size_t i=0; i<nmodes+1; ++i) {
    result = hipFree(tmp_mats[i]);
    spt_CheckCudaError(result != 0, "CUDA SpTns MTTKRP");
  }
  free(tmp_mats);
  result = hipFree(dev_mats);
  spt_CheckCudaError(result != 0, "CUDA SpTns MTTKRP");

  spt_SparseTensorFreeAllSplits(splits);


  return 0;
}


